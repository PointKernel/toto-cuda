#include "hip/hip_runtime.h"
#include <hip/hip_runtime_api.h>
#include <iostream>

__global__ void emptyKernel() {}

int main()
{
  const int N = 100000;
  float time, total = 0.f;

  hipEvent_t start, stop;
  hipEventCreate(&start);
  hipEventCreate(&stop);

  for (int j = 1; j <= 4096; j *= 2) {
    for (int k = 1; k <= 1024; k *= 2) {
      for (int i = 0; i < N; i++) {
        hipEventRecord(start, 0);
        emptyKernel<<<1, 1>>>();
        hipEventRecord(stop, 0);
        hipEventSynchronize(stop);
        hipEventElapsedTime(&time, start, stop);
        total = total + time;
      }
      std::cout << "Kernel: " << j << "X" << k << "\tlaunch overhead: " << total / N * 1000
                << " us\n";
      total = 0.f;
    }
  }

  total = 0.f;

  void* dst = nullptr;
  void* src = nullptr;
  for (int i = 0; i < N; i++) {
    hipEventRecord(start, 0);
    checkCudaErrors(hipMemcpy(dst, src, 0, hipMemcpyDefault));
    hipEventRecord(stop, 0);
    hipEventSynchronize(stop);
    hipEventElapsedTime(&time, start, stop);
    total = total + time;
  }

  std::cout << "\nData transfer overhead: " << total / N * 1000 << " us\n";

  return 0;
}
