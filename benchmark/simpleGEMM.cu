#include "hip/hip_runtime.h"
#include <cstdlib>
#include <iostream>

#include <hipblas.h>
#include <hip/hip_fp16.h>
#include <hip/hip_runtime_api.h>

using namespace std;

void init(half *A, half *B, float *C, size_t m, size_t n, size_t k) {
  for (size_t i = 0; i < m; i++)
    for (size_t j = 0; j < k; j++)
      A[i * k + j] = __float2half(static_cast<float>(rand() % 100));
  for (size_t i = 0; i < k; i++)
    for (size_t j = 0; j < n; j++)
      B[i * n + j] = __float2half(static_cast<float>(rand() % 100));
  for (size_t i = 0; i < m; i++)
    for (size_t j = 0; j < n; j++)
      C[i * n + j] = static_cast<float>(rand() % 100);
}

int main(int argc, char *argv[]) {
  size_t m_global, n_global, k_global;
  if (argc == 2) {
    cout << "\nMatrix size: " << argv[1] << endl;
    m_global = n_global = k_global = atoi(argv[1]);
  } else {
    const size_t size = 4096;
    cout << "\nUsing default matrix size: " << size << endl;
    m_global = n_global = k_global = size;
  }

  // declare host data
  half *A_h;
  half *B_h;
  float *C_h;
  A_h = (half *)malloc(m_global * k_global * sizeof(half));
  B_h = (half *)malloc(k_global * n_global * sizeof(half));
  C_h = (float *)malloc(m_global * n_global * sizeof(float));

  // declare device data
  half *A_d;
  half *B_d;
  float *C_d;
  checkCudaErrors(hipMalloc(reinterpret_cast<void **>(&A_d),
                             m_global * k_global * sizeof(half)));
  checkCudaErrors(hipMalloc(reinterpret_cast<void **>(&B_d),
                             k_global * n_global * sizeof(half)));
  checkCudaErrors(hipMalloc(reinterpret_cast<void **>(&C_d),
                             m_global * n_global * sizeof(float)));

  // initialize host data
  init(A_h, B_h, C_h, m_global, n_global, k_global);

  // copy host data to device
  hipMemcpy(reinterpret_cast<void *>(A_d), reinterpret_cast<void *>(A_h),
             m_global * k_global, hipMemcpyHostToDevice);
  hipMemcpy(reinterpret_cast<void *>(B_d), reinterpret_cast<void *>(B_h),
             k_global * n_global, hipMemcpyHostToDevice);
  hipMemcpy(reinterpret_cast<void *>(C_d), reinterpret_cast<void *>(C_h),
             m_global * n_global, hipMemcpyHostToDevice);
  hipDeviceSynchronize();

  // create CUDA events for timing measurement
  hipEvent_t start, stop;
  hipEventCreate(&start);
  hipEventCreate(&stop);

  hipblasHandle_t cublasHandle;
  hipblasCreate(&cublasHandle);
  hipblasSetMathMode(cublasHandle, HIPBLAS_TENSOR_OP_MATH);

  float alpha = 2.0f;
  float beta = 2.0f;

  // dim3 gridDim;
  // dim3 blockDim;
  for (int i = 0; i < 10; i++) {
    hipEventRecord(start);
    checkCudaErrors(hipblasGemmEx(
        cublasHandle, HIPBLAS_OP_N, HIPBLAS_OP_N, m_global, n_global, k_global,
        &alpha, A_d, HIP_R_16F, m_global, B_d, HIP_R_16F, k_global, &beta,
        C_d, HIP_R_32F, m_global, HIP_R_32F, CUBLAS_GEMM_DFALT_TENSOR_OP));
    hipEventRecord(stop);
  }

  hipMemcpy(reinterpret_cast<void *>(C_h), reinterpret_cast<void *>(C_d),
             m_global * n_global, hipMemcpyDeviceToHost);
  hipDeviceSynchronize();

  // print kernel runtime
  hipEventSynchronize(stop);
  float milliseconds = 0.f;
  hipEventElapsedTime(&milliseconds, start, stop);
  double seconds = static_cast<double>(milliseconds) / 1000.;
  cout << "runtime: " << seconds << endl;
  cout << "Tensor TFLOPS: "
       << (m_global * n_global * k_global) * 2.0 / seconds / 1e12 << endl;

  // free the allocated memory
  free(A_h);
  free(B_h);
  free(C_h);
  hipFree(reinterpret_cast<void *>(A_d));
  hipFree(reinterpret_cast<void *>(B_d));
  hipFree(reinterpret_cast<void *>(C_d));

  return 0;
}
