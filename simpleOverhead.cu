#include "hip/hip_runtime.h"
#include <hip/hip_runtime_api.h>
#include <iostream>

__global__ void emptyKernel() {}

int main() {
  const int N = 100000;
  float time, total = 0.f;

  hipEvent_t start, stop;
  hipEventCreate(&start);
  hipEventCreate(&stop);

  for (int i = 0; i < N; i++) {
    hipEventRecord(start, 0);
    emptyKernel<<<1, 1>>>();
    hipEventRecord(stop, 0);
    hipEventSynchronize(stop);
    hipEventElapsedTime(&time, start, stop);
    total = total + time;
  }

  std::cout << "Kernel launch overhead: " << total / N * 1000 << " us\n";

  total = 0.f;

  void *dst = nullptr;
  void *src = nullptr;
  for (int i = 0; i < N; i++) {
    hipEventRecord(start, 0);
    checkCudaErrors(hipMemcpy(dst, src, 0, hipMemcpyDefault));
    hipEventRecord(stop, 0);
    hipEventSynchronize(stop);
    hipEventElapsedTime(&time, start, stop);
    total = total + time;
  }

  std::cout << "Data transfer overhead: " << total / N * 1000 << " us\n";

  return 0;
}
