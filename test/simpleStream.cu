
#include <hip/hip_runtime.h>
#include <iostream>

#include <assert.h>

int main()
{
  hipStream_t stream;

  hipStreamCreate(&stream);

  assert(hipStreamDestroy(stream) == 0 /* destroyed without issues*/);
  assert(hipStreamDestroy(stream) == 709 /* cudaErrorContextIsDestroyed */);

  // Default stream is non-owning thus should not be destroyed by users
  assert(hipStreamDestroy(0 /*default stream*/) == 400 /* cudaErrorInvalidResourceHandle */);

  return 0;
}
