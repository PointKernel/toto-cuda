
#include <hip/hip_runtime.h>
#include <iostream>

int main() {
  hipStream_t stream;

  hipStreamCreate(&stream);

  std::cout << hipStreamDestroy(stream) << "\n";
  std::cout << hipStreamDestroy(stream) << "\n";
  std::cout << hipStreamDestroy(stream) << "\n";

  std::cout << hipStreamDestroy(hipStreamDefault) << "\n";

  return 0;
}
