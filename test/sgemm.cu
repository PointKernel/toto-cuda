
#include <hip/hip_runtime.h>
#include <iostream>

const long int IMAGE_SIZE = 4096;
const int BLOCK_SIZE = 16;

const float alpha = 2.f;
const float beta = 2.f;

__global__
void sgemmNaive(float *A, float *B, float *C, int N) {
  int row = blockIdx.y * blockDim.y + threadIdx.y;
  int col = blockIdx.x * blockDim.x + threadIdx.x;

  float val = 0.f;
  for (int i  = 0; i < N; i++)
    val += A[row * N + i] * B[i * N + col];
  C[row * N + col] = alpha * val + beta * C[row * N + col];
}

__global__
void sgemmSHM(float *A, float *B, float *C, int N) {
  int row = blockIdx.y * blockDim.y + threadIdx.y;
  int col = blockIdx.x * blockDim.x + threadIdx.x;

  float val = 0.f;

  __shared__ float As[BLOCK_SIZE][BLOCK_SIZE];
  __shared__ float Bs[BLOCK_SIZE][BLOCK_SIZE];

  for (int i  = 0; i < N / BLOCK_SIZE; i++) {
    As[threadIdx.y][threadIdx.x] = A[row * N + BLOCK_SIZE * i + threadIdx.x];
    Bs[threadIdx.y][threadIdx.x] = B[(threadIdx.y + i * BLOCK_SIZE) * N + col];
    __syncthreads();
    for (int j = 0; j < BLOCK_SIZE; j++)
      val += As[threadIdx.y][j] * Bs[j][threadIdx.x];
    __syncthreads();
  }
  C[row * N + col] = alpha * val + beta * C[row * N + col];
}

int main() {
  float *A, *A_d, *B, *B_d, *C, *C_d;
  const int data_size = IMAGE_SIZE * IMAGE_SIZE * sizeof(float);

  hipHostMalloc(&A, data_size, hipHostMallocDefault);
  hipHostMalloc(&B, data_size, hipHostMallocDefault);
  hipHostMalloc(&C, data_size, hipHostMallocDefault);
  hipMalloc(&A_d, data_size);
  hipMalloc(&B_d, data_size);
  hipMalloc(&C_d, data_size);

  const int grid_size = IMAGE_SIZE / BLOCK_SIZE;
  dim3 grid(grid_size, grid_size);  // 128 * 128
  dim3 block(BLOCK_SIZE, BLOCK_SIZE); // 32 x 32 = 1024


  for (int i = 0; i < IMAGE_SIZE * IMAGE_SIZE; i ++) {
    A[i] = 1.f;
    B[i] = 1.f;
    C[i] = 1.f;
  }

  hipEvent_t start, stop;
  hipEventCreate(&start);
  hipEventCreate(&stop);

  hipMemcpy(A_d, A, data_size, hipMemcpyHostToDevice);
  hipMemcpy(B_d, B, data_size, hipMemcpyHostToDevice);
  hipMemcpy(C_d, C, data_size, hipMemcpyHostToDevice);

  hipEventRecord(start);
  sgemmNaive<<<grid, block>>>(A_d, B_d, C_d, IMAGE_SIZE);
  hipEventRecord(stop);

  hipMemcpy(C, C_d, data_size, hipMemcpyDeviceToHost);
  hipEventSynchronize(stop);

  // runtime and FLOP rate
  float milliseconds = 0.f;
  hipEventElapsedTime(&milliseconds, start, stop);
  double seconds = static_cast<double>(milliseconds) / 1000.;
  std::cout << "sgemmNaive runtime: " << seconds << "\n";
  std::cout << "Performance (TFLOPS/s): "
       << (IMAGE_SIZE * IMAGE_SIZE * IMAGE_SIZE) * 2.0 / seconds / 1e12 << "\n\n";

  for (int i = 0; i < IMAGE_SIZE * IMAGE_SIZE; i ++) {
    A[i] = 1.f;
    B[i] = 1.f;
    C[i] = 1.f;
  }

  hipEvent_t start1, stop1;
  hipEventCreate(&start1);
  hipEventCreate(&stop1);

  hipMemcpy(A_d, A, data_size, hipMemcpyHostToDevice);
  hipMemcpy(B_d, B, data_size, hipMemcpyHostToDevice);
  hipMemcpy(C_d, C, data_size, hipMemcpyHostToDevice);

  hipEventRecord(start1);
  sgemmSHM<<<grid, block>>>(A_d, B_d, C_d, IMAGE_SIZE);
  hipEventRecord(stop1);

  hipMemcpy(C, C_d, data_size, hipMemcpyDeviceToHost);
  hipEventSynchronize(stop1);

  // runtime and FLOP rate
  milliseconds = 0.f;
  hipEventElapsedTime(&milliseconds, start1, stop1);
  seconds = static_cast<double>(milliseconds) / 1000.;
  std::cout << "sgemmSHM runtime: " << seconds << "\n";
  std::cout << "Performance (TFLOPS/s): "
       << (IMAGE_SIZE * IMAGE_SIZE * IMAGE_SIZE) * 2.0 / seconds / 1e12 << "\n\n";

  hipFree(A_d);
  hipFree(B_d);
  hipFree(C_d);

  hipHostFree(A);
  hipHostFree(B);
  hipHostFree(C);
}
