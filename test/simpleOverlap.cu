#include "hip/hip_runtime.h"
#include <iostream>

#include <hip/hip_runtime_api.h>

using namespace std;

__global__ void kernel(float *a, int offset) {
  int i = offset + threadIdx.x + blockIdx.x * blockDim.x;
  float x = (float)i;
  float s = sinf(x);
  float c = cosf(x);
  a[i] = a[i] + sqrtf(s * s + c * c);
}

int main(int argc, char **argv) {
  const int blockSize = 256, nStreams = 4;
  const int n = 1024;
  const int streamSize = n / nStreams;
  const int streamBytes = streamSize * sizeof(float);
  const int bytes = n * sizeof(float);

  // allocate pinned host memory and device memory
  float *a, *d_a;
  checkCudaErrors(hipHostMalloc((void **)&a, bytes)); // host pinned
  checkCudaErrors(hipMalloc((void **)&d_a, bytes));   // device

  // create events and streams
  hipEvent_t start, stop;
  hipStream_t stream[nStreams];
  checkCudaErrors(hipEventCreate(&start));
  checkCudaErrors(hipEventCreate(&stop));

  for (int i = 0; i < nStreams; ++i)
    checkCudaErrors(hipStreamCreate(&stream[i]));

  memset(a, 0, bytes);

  checkCudaErrors(hipEventRecord(start));
  for (int i = 0; i < nStreams; ++i) {
    int offset = i * streamSize;
    checkCudaErrors(hipMemcpyAsync(&d_a[offset], &a[offset], streamBytes,
                                    hipMemcpyHostToDevice, stream[i]));
    kernel<<<streamSize / blockSize, blockSize, 0, stream[i]>>>(d_a, offset);
    checkCudaErrors(hipMemcpyAsync(&a[offset], &d_a[offset], streamBytes,
                                    hipMemcpyDeviceToHost, stream[i]));
  }
  checkCudaErrors(hipEventRecord(stop));
  checkCudaErrors(hipEventSynchronize(stop));

  float milliseconds = 0.f;
  checkCudaErrors(hipEventElapsedTime(&milliseconds, start, stop));

  cout << "Runtime: " << milliseconds << " ms\n";

  // cleanup
  checkCudaErrors(hipEventDestroy(start));
  checkCudaErrors(hipEventDestroy(stop));
  for (int i = 0; i < nStreams; ++i)
    checkCudaErrors(hipStreamDestroy(stream[i]));
  hipFree(d_a);
  hipHostFree(a);

  return 0;
}
