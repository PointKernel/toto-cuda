
#include <hip/hip_runtime.h>
#include <iostream>

const int image_size  = 4096;
const int filter_size = 3;

__global__ void conv2d(int* A, int* B, int* C, int N, int n)
{
  int row = blockIdx.y * blockDim.y + threadIdx.y;
  int col = blockIdx.x * blockDim.x + threadIdx.x;

  const int offset = n / 2;

  int row_i = threadIdx.y - offset;
  int col_i = threadIdx.x - offset;

  __shared__ int shm[16][16];

  shm[threadIdx.y][threadIdx.x] = A[row * N + col];

  __syncthreads();

  int val = 0;

  for (int i = 0; i < n; i++)
    for (int j = 0; j < n; j++)
      if ((0 <= (i + col_i) && (i + col_i) < 16))
        if ((0 <= (j + row_i) && (j + row_i) < 16)) val += shm[j + row_i][i + col_i] * C[j * n + i];

  B[row * N + col] = val;
}

int main()
{
  int *A, *A_d, *B, *B_d, *C, *C_d;
  const int data_size   = image_size * image_size * sizeof(int);
  const int kernel_size = filter_size * filter_size * sizeof(int);

  hipHostMalloc(&A, data_size, hipHostMallocDefault);
  hipHostMalloc(&B, data_size, hipHostMallocDefault);
  hipHostMalloc(&C, kernel_size, hipHostMallocDefault);

  for (int i = 0; i < image_size * image_size; i++)
    A[i] = 1;
  memset(B, 0, data_size);
  for (int i = 0; i < filter_size * filter_size; i++)
    C[i] = 2;

  hipMalloc(&A_d, data_size);
  hipMalloc(&B_d, data_size);
  hipMalloc(&C_d, kernel_size);

  hipEvent_t start, stop;
  hipEventCreate(&start);
  hipEventCreate(&stop);

  hipMemcpy(A_d, A, data_size, hipMemcpyHostToDevice);
  hipMemcpy(B_d, B, data_size, hipMemcpyHostToDevice);
  hipMemcpy(C_d, C, kernel_size, hipMemcpyHostToDevice);

  const int block_size = 16;
  const int grid_size  = (image_size + block_size - 1) / block_size;
  dim3 grid(grid_size, grid_size);
  dim3 block(block_size, block_size);

  hipEventRecord(start);
  conv2d<<<grid, block>>>(A_d, B_d, C_d, image_size, filter_size);
  hipEventRecord(stop);

  hipMemcpy(B, B_d, data_size, hipMemcpyDeviceToHost);

  hipEventSynchronize(stop);
  float milliseconds = 0;
  hipEventElapsedTime(&milliseconds, start, stop);

  for (int i = 0; i < 10; i++) {
    for (int j = 0; j < 10; j++)
      std::cout << B[i * image_size + j] << " ";
    std::cout << "\n";
  }
  std::cout << "Kernel run time: " << milliseconds << " ms\n";

  hipFree(A_d);
  hipFree(B_d);
  hipFree(C_d);

  hipHostFree(A);
  hipHostFree(B);
  hipHostFree(C);
}
